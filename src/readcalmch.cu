#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : readcalmch.cu
 Author      : Ting-Wen Yu
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <fstream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define BLOCK_SIZE 512

using namespace std;

typedef struct _MCHInfo {
	char	fname_mch[30];
	char	fname_inp[30];
	char	magicheader[4];
	unsigned int	version, maxmedia, detnum, colcount, totalphoton, detected, savedphoton, seedbyte;
	unsigned int 	junk[5];
	float 	unitmm, normalizer;
	float 	na, n0, theta; 	// load from .inp
	float 	*mua;			// load from .inp
	unsigned int sizeOfRawData, sizeOfData, sizeOfResult;
}MCHInfo;

typedef struct _MCHData {
	float 	*rawdata;		//array length: sizeOfRawData
	int		*detid;			//array length:	sizeOfData
	float 	*weight;		//array length: sizeOfData
	float 	*result;		//array length: sizeOfResult
}MCHData;

template <typename T>
void arraymapping_1d(T *origin, T *copy, unsigned int size){
	for (unsigned int i = 0; i < size; ++i)
		copy[i] = origin[i];
}

template <typename T>
void fprintf1DArray(char fname[], T *data, unsigned int size)
{
	printf("Print to %s ...\n",fname);
	ofstream myfile;

	myfile.open(fname);
	for (unsigned int i = 0; i < size; ++i)
		myfile << "[" << i << "]\t" << data[i] << endl;

	myfile.close();
}

void initloadpara(MCHInfo *info, MCHData *data){

	FILE *fptr_mch, *fptr_inp;

	// set constant
	info->n0 = 1.457; 	printf("n0\t\t%f\n",info->n0);
	info->na = 0.22; 	printf("na\t\t%f\n",info->na);
	info->theta = asin(info->na/info->n0); printf("theta\t\t%f\n",info->theta);

	// specify .mch fname
	printf("Enter .mch file name:");
	scanf("%s",&(info->fname_mch));
	printf("Loading from %s ...\n",info->fname_mch);

	// load from fptr_mch
	fptr_mch = fopen(info->fname_mch,"rb");

	fread(info->magicheader,sizeof(char),4,fptr_mch);				printf("version\t\t%c%c%c%c\n",info->magicheader[0],info->magicheader[1],info->magicheader[2],info->magicheader[3]);
	fread(&(info->version),sizeof(unsigned int),1,fptr_mch);		printf("version\t\t%d\n",info->version);
	fread(&(info->maxmedia),sizeof(unsigned int),1,fptr_mch);		printf("mexmedia\t%d\n",info->maxmedia);
	fread(&(info->detnum),sizeof(unsigned int),1,fptr_mch);			printf("detnum\t\t%d\n",info->detnum);
	fread(&(info->colcount),sizeof(unsigned int),1,fptr_mch);		printf("colcount\t%d\n",info->colcount);
	fread(&(info->totalphoton),sizeof(unsigned int),1,fptr_mch);	printf("totalphoton\t%d\n",info->totalphoton);
	fread(&(info->detected),sizeof(unsigned int),1,fptr_mch);		printf("detected\t%d\n",info->detected);
	fread(&(info->savedphoton),sizeof(unsigned int),1,fptr_mch);	printf("savedphoton\t%d\n",info->savedphoton);
	fread(&(info->unitmm),sizeof(float),1,fptr_mch);				printf("unitmm\t\t%f\n",info->unitmm);
	fread(&(info->seedbyte),sizeof(unsigned int),1,fptr_mch);		printf("seedbyte\t%d\n",info->seedbyte);
	fread(&(info->normalizer),sizeof(float),1,fptr_mch);			printf("normalizer\t%f\n",info->normalizer);
	fread(info->junk,sizeof(unsigned int),5,fptr_mch);				printf("junk\t\t%d%d%d%d%d\n",info->junk[0],info->junk[1],info->junk[2],info->junk[3],info->junk[4]);

	//allocate memory
	info->sizeOfData = info->savedphoton;
	data->detid = (int*) malloc (sizeof(int)*info->sizeOfData);
	data->weight = (float*) malloc (sizeof(float)*info->sizeOfData);

	info->sizeOfResult = info->detnum;
	data->result = (float*) malloc (sizeof(float)*info->sizeOfResult);

	info->sizeOfRawData = info->savedphoton*info->colcount;
	data->rawdata = (float*) malloc (sizeof(float)*info->sizeOfRawData);
	fread(data->rawdata ,sizeof(float), info->sizeOfRawData,fptr_mch); /* did not scaled back to 1 mm yet */


	// specify .inp fname
	printf("Enter .inp file name:");
	scanf("%s",&(info->fname_inp));
	printf("Loading from %s ...\n",info->fname_inp);

	// load from fptr_inp
	fptr_inp = fopen(info->fname_inp,"r");
	char junkc[50];
	for (int i = 0; i < 10; ++i)
		fgets(junkc, 50, fptr_inp); //discard from line 1 to 10

	unsigned int sizeOfMua = info->maxmedia;
	double junkf1, junkf2, junkf3, junkf4;
	info->mua = (float*) malloc (sizeof(float)*sizeOfMua);
	for(int i = 0; i < sizeOfMua; ++i)
	{
		printf("mua %d:",i);
		fscanf(fptr_inp,"%lf %lf %lf %lf",&(junkf1), &(junkf2), &(junkf3), &(junkf4));
		info->mua[i] = (float)junkf3; //casting double into float, and stored in mua[i]
		printf("\t%e\n",info->mua[i]);
	}

	// close
	fclose(fptr_mch);
	fclose(fptr_inp);

}
/**
 * CUDA kernel that computes reflectance values for each photon
 */
__global__ void calRefPerPhotonKernel(unsigned int size, unsigned int colcount, unsigned int maxmedia, float *rawdata, int *detid, float *weight, float *mua, float unitmm, float theta) {

	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x; //i.e. rowcount

	if (idx < size){
		detid[idx] = (int)rawdata[idx*colcount];
		weight[idx] = 0.0;

		float temp = 0.0;
		if (acosf(abs(rawdata[(idx+1)*colcount-1])) <= theta){
			for (unsigned int i = 0; i < maxmedia; ++i)
				temp += (-1.0)*unitmm*mua[i]*rawdata[idx*colcount + (2+i)];
			weight[idx] = __expf(temp);
		}
	}
}
void calref_photon(MCHInfo *info,MCHData *data){

	float *gRawdata, *gWeight, *gMua;
	int *gDetid;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gRawdata, sizeof(float)*info->sizeOfRawData));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gDetid, sizeof(int)*info->sizeOfData));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gWeight, sizeof(float)*info->sizeOfData));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gMua, sizeof(float)*info->maxmedia));

	CUDA_CHECK_RETURN(hipMemcpy(gRawdata, data->rawdata, sizeof(float)*info->sizeOfRawData, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gMua, info->mua, sizeof(float)*info->maxmedia, hipMemcpyHostToDevice));

	unsigned int blockCount = (info->sizeOfData + BLOCK_SIZE-1)/BLOCK_SIZE;
	calRefPerPhotonKernel<<<blockCount, BLOCK_SIZE>>> (info->sizeOfData, info->colcount, info->maxmedia, gRawdata, gDetid, gWeight, gMua, info->unitmm, info->theta);

	CUDA_CHECK_RETURN(hipMemcpy(data->detid, gDetid, sizeof(int)*info->sizeOfData, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(data->weight, gWeight, sizeof(float)*info->sizeOfData, hipMemcpyDeviceToHost));


	CUDA_CHECK_RETURN(hipFree(gRawdata));
	CUDA_CHECK_RETURN(hipFree(gDetid));
	CUDA_CHECK_RETURN(hipFree(gWeight));
	CUDA_CHECK_RETURN(hipFree(gMua));

}

void sortbykey(MCHInfo *info, MCHData *data){

	//copy values from pointer to static array
	int keys[info->sizeOfData];
	arraymapping_1d<int>(data->detid, keys, info->sizeOfData);

	float values[info->sizeOfData];
	arraymapping_1d<float>(data->weight, values, info->sizeOfData);

	//call main function
	const int N = info->sizeOfData;
	thrust::sort_by_key(thrust::host, keys, keys + N, values);

	//copy values from static array to pointer
	arraymapping_1d<int >(keys, data->detid, info->sizeOfData);
	arraymapping_1d<float>(values, data->weight, info->sizeOfData);


}
void calref_det(MCHInfo *info, MCHData *data){

	//copy values from pointer to static array
	int keysIn[info->sizeOfData];
	arraymapping_1d<int>(data->detid, keysIn, info->sizeOfData);

	float valuesIn[info->sizeOfData];
	arraymapping_1d<float>(data->weight, valuesIn, info->sizeOfData);

	int keysOut[info->sizeOfResult];

	float valuesOut[info->sizeOfResult];

	//call main function
	const int N = info->sizeOfData;
	thrust::reduce_by_key(thrust::host, keysIn, keysIn + N, valuesIn, keysOut, valuesOut);

	//copy values from static array to pointer
	arraymapping_1d<float>(valuesOut, data->result, info->sizeOfResult);

}
void printresult(MCHInfo *info, MCHData *data){

	// print result
	char f1[] = "result.txt";
	fprintf1DArray(f1, data->result, info->sizeOfResult);

	// print result./totalphoton
	double temp[info->sizeOfResult];
	for (unsigned i = 0; i < info->sizeOfResult; ++i)
		temp[i] = data->result[i]/info->totalphoton;
	char f2[] = "result_dividedTotalPhoton.txt";
	fprintf1DArray(f2, temp, info->sizeOfResult);

}
void clearmch(MCHInfo *info, MCHData *data){
	if(info->mua){
		free(info->mua);
		info->mua = NULL;
	}
	if(data->rawdata){
		free(data->rawdata);
		data->rawdata = NULL;
	}
	if(data->detid){
		free(data->detid);
		data->detid = NULL;
	}
	if(data->weight){
		free(data->weight);
		data->weight = NULL;
	}
	if(data->result){
		free(data->result);
		data->result = NULL;
	}
}

int main(void)
{
	MCHInfo info;
	MCHData data;

	initloadpara(&info,&data);
	calref_photon(&info,&data);
	sortbykey(&info,&data);
	calref_det(&info,&data);
	printresult(&info,&data);
	clearmch(&info,&data);

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

