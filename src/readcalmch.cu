/*
 ============================================================================
 Name        : readcalmch.cu
 Author      : Ting-Wen Yu
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

void initialize1DArray(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = 0.0;
}

void initialize2DArray(float **data2D, unsigned *size_2D)
{
	unsigned height = size_2D[0];
	unsigned width = size_2D[1];

	for (unsigned i = 0; i < height; ++i){
		for (unsigned j = 0; j < width; ++j)
			data2D[i][j] = 0.0;
	}
}

void fprintf1DArray(float *data, unsigned size)
{
	FILE *fptr = fopen("1DArray.txt","w");
	for (unsigned i = 0; i < size; ++i)
		fprintf(fptr,"[%d]\t%f\n",i,data[i]);
	fclose(fptr);
}

void fprintf2DArray(float **data2D, unsigned *size_2D)
{
	unsigned height = size_2D[0];
	unsigned width = size_2D[1];

	FILE *fptr = fopen("2DArray.txt","w");
	for (unsigned i = 0; i < height; ++i){
		for (unsigned j = 0; j < width; ++j)
			fprintf(fptr,"[%d][%d]\t%f\n",i,j,data2D[i][j]);
	}
	fclose(fptr);
}

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = data[idx]+idx;
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuCalReflectance(float *data, unsigned *size_2D)
{
	unsigned size = size_2D[0];
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

int main(void)
{
	/* 1D */
	int WORK_SIZE = 100;
	float *data = new float[WORK_SIZE];
	initialize1DArray (data, WORK_SIZE);

	float *recGpu = gpuReciprocal(data, WORK_SIZE);
	fprintf1DArray (recGpu, WORK_SIZE);

	/* 2D */
	unsigned *WORK_SIZE_2D = new unsigned[2];
	WORK_SIZE_2D[0] = 100; /* array height */
	WORK_SIZE_2D[1] = 2;   /* array width */

	float **data2D = new float*[WORK_SIZE_2D[0]];
	for(int i = 0; i < WORK_SIZE_2D[0]; ++i)
		data2D[i] = new float[WORK_SIZE_2D[1]];
	initialize2DArray (data2D, WORK_SIZE_2D);
	fprintf2DArray (data2D, WORK_SIZE_2D);

	float *refPerDet = gpuCalReflectance(data, WORK_SIZE_2D);



	/* Sum up in host */
	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);
	float result = std::accumulate (refPerDet, refPerDet+WORK_SIZE, 0.0);

	/* Verify the results */
	std::cout<<"gpuSum = "<<gpuSum<<std::endl;
	std::cout<<"result = "<<result<<std::endl;

	/* Free memory */
	delete[] data;
	delete[] recGpu;
	for(int i = 0; i < WORK_SIZE_2D[0]; ++i)
		delete[] data2D[i];
	delete[] data2D;
	delete[] WORK_SIZE_2D;
	delete[] refPerDet;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

